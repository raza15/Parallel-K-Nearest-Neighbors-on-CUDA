
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10

__global__ 
void outputFromGPU() {
	printf("Hello from GPU!\n");
}

__global__
void add(int a, int b, int * c) {
	*c = a + b;
}

__global__
void addTwoArrays(int* a, int* b, int* c) {
	int bid = blockIdx.x;
	if(bid < N) {
		c[bid] = a[bid] + b[bid];
	}
}

void mainForAdd() {
	// printf("Hello from CPU!\n");
        // outputFromGPU<<<2,5>>>();
        // cudaDeviceSynchronize();
	int a, b, c;
        int * dev_c;
        a = 3;
        b = 4;
        hipMalloc((void **) &dev_c, sizeof(int));
        add<<<1,1>>>(a,b,dev_c);
        hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
        printf("%d + %d = %d\n", a, b, c);
        hipFree(dev_c);
}

void mainForAddTwoArrays() {
	int i, a[N], b[N], c[N];
	int *dev_a; 
	int *dev_b; 
	int *dev_c;

	hipMalloc((void**) &dev_a, N*sizeof(int));
	hipMalloc((void**) &dev_b, N*sizeof(int));
	hipMalloc((void**) &dev_c, N*sizeof(int));
	
	for(i = 0; i < N; i++) {
		a[i] = i;
		b[i] = i*i;
	}
	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
	// printf("here1\n");
	addTwoArrays<<<N, 1>>>(dev_a, dev_b, dev_c);
	// printf("here2\n");
	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
	
	printf("\na + b = c\n");
	for(i = 0; i < N; i++) {
		printf("\n%5d + %5d = %5d\n", a[i], b[i], c[i]);
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

int main(void) {
	mainForAddTwoArrays();
}
