
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>

#define MAX_VALUE 2147483647
#define numThreads 32

void printMatrix(int *matrix, int users, int attributes) {
	// printf("Matrix:-\n");
	for(int i = 0; i < (users * attributes); i++) {
		if(i % attributes == 0 && i != 0) {
			printf("\n%d ", matrix[i]);
		} else {
			printf("%d ", matrix[i]);
		}
	}
	printf("\n");
}

int checker(char* input, char* check) {
    int i,result=1;
    for(i=0; input[i]!='\0' || check[i]!='\0'; i++) {
        if(input[i] != check[i]) {
            result=0;
            break;
        }
    }
    return result;
}

void preliminarySteps(int argc, char** argv, int** dataSetPtr, int** scoresPtr, int* usersPtr, int* attributesPtr, int* kPtr) {
    // Check input
    if(argc < 5) {
        printf("Usage: %s <k> <users> <attributes> <serial/parallel>\n", argv[0]);
        exit(0);
    }
	int * dataSet;
	int k = atoi(argv[1]);
	int users = atoi(argv[2]);
	int attributes = atoi(argv[3]);
	*usersPtr = users;
	*attributesPtr = attributes;
	*kPtr = k;

	dataSet = (int*) malloc(sizeof(int) * users * attributes);
    if(dataSet != NULL) {
//        printf("Allocated an array for %d users and %d attributes\n", users, attributes);
    } else {
        printf("Couldn't allocate dataSet array, quitting!\n");
        exit(0);
    }

    // Seed the RNG
    srand(time(NULL));
    // Now fill dataSet with some values
    for(int i=0; i < (users * attributes); i++)
        //dataSet[i] = rand() % 1000; // Random integers between 0 and 1,000
        dataSet[i] = rand() % 15; // Random integers between 0 and 1,000
    *dataSetPtr = dataSet;
    int *scores;
    scores = (int *)malloc(sizeof(int) * users * users);
    if(scores != NULL) {
  //      printf("Allocated a square scores array for %d users\n", users);
    } else {
        printf("Couldn't allocate scores array, quitting!\n");
        exit(0);
    }
    *scoresPtr = scores;
}

void calculateScore(int* matrix, int* scores, int users, int attributes, int user1, int user2) {
	int answer = 0;
	int user1Start = attributes*user1;
	int user1End = user1Start + attributes - 1;
	int user2Start = attributes*user2;
	int user2End = user2Start + attributes - 1;
	
	int i; int j; int difference;
	for(i = user1Start, j = user2Start; i <= user1End && j <= user2End ; i++, j++) {
		difference = matrix[i] - matrix[j];
		answer += difference*difference;
	}
	
	scores[user1*users + user2] = answer;
}

void calculateScores(int *matrix, int *scores, int users, int attributes) {
	int user1; int user2;
	for(user1 = 0; user1 < users; user1++) {
		for(user2 = 0; user2 < users; user2++) {
			calculateScore(matrix, scores, users, attributes, user1, user2);
		}
	}
}

__global__ void calculateScoreKernel(int *matrix, int *scores, int users, int attributes) {
        int user1 = numThreads*blockIdx.x + threadIdx.x;
	int user2 = numThreads*blockIdx.y + threadIdx.y;

	if(user1 >= 0 && user1 < users && user2 >= 0 && user2 < users) {
        int answer = 0;
        int user1Start = attributes*user1;
        int user1End = user1Start + attributes - 1;
        int user2Start = attributes*user2;
        int user2End = user2Start + attributes - 1;

        int i; int j; int difference;
        for(i = user1Start, j = user2Start; i <= user1End && j <= user2End ; i++, j++) {
                difference = matrix[i] - matrix[j];
                answer += difference*difference;
        }
	/*
	# if __CUDA_ARCH__>=200
                printf("%d, %d, %d, %d => %d \n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, answer);
        #endif
	*/
        scores[user1*users + user2] = answer;
	}
}

void launchCalculateScoreKernel(int * dataSet, int * scores, int users, int attributes) {
	int * dev_dataSet;
	int * dev_scores;
	
	hipMalloc((void**) &dev_dataSet, users*attributes*sizeof(int));
	hipMalloc((void**) &dev_scores, users*users*sizeof(int));
	
	hipMemcpy(dev_dataSet, dataSet, users*attributes*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_scores, scores, users*users*sizeof(int), hipMemcpyHostToDevice);

	int numBlocks = (int) ceil(users*1.0/numThreads);
	dim3 grid( numBlocks, numBlocks, 1 );
	dim3 block( numThreads, numThreads, 1 );
	calculateScoreKernel<<< grid, block >>>(dev_dataSet, dev_scores, users, attributes);

	hipMemcpy(scores, dev_scores, users*users*sizeof(int), hipMemcpyDeviceToHost);
}

__global__ void calculateKNearestKernel(int * scores, int * kNearest, int users, int K) {
		int minValue, minIndex, value, user, k, index;
		user = numThreads*blockIdx.x + threadIdx.x;
                for(k = 0; k < K; k++) {
                        minValue = MAX_VALUE;
                        minIndex = -1;
                        for(index = 0; index < users; index++) {
                                value = scores[user*users + index];
                                if(value < minValue && index != user) {
                                        minValue = value;
                                        minIndex = index;
                                }
                        }
                        if(minIndex != -1) {
                                // scores[user*users + minIndex] = MAX_VALUE;
                        }
                        // kNearest[user*users + k] = minIndex;
                }
}

// arguments: scores, kNearest, users, k
void launchCalculateKNearestKernel(int * dataSet, int * scores, int users, int k) {
	int * dev_dataSet;
        int * dev_scores;

        hipMalloc((void**) &dev_dataSet, users*users*sizeof(int));
        hipMalloc((void**) &dev_scores, users*k*sizeof(int));

        hipMemcpy(dev_dataSet, dataSet, users*users*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_scores, scores, users*k*sizeof(int), hipMemcpyHostToDevice);

        int numBlocks = (int) ceil(users*1.0/numThreads);
        dim3 grid( numBlocks, numBlocks, 1 );
        dim3 block( numThreads, numThreads, 1 );
        calculateKNearestKernel<<< grid, block >>>(dev_dataSet, dev_scores, users, k);

        hipMemcpy(scores, dev_scores, users*k*sizeof(int), hipMemcpyDeviceToHost);
}

void writeToFile(clock_t start, clock_t end, char * whichProgramToRun, int users, int attributes, int k, char * fileName) {
	FILE * file;
	if(checker(whichProgramToRun, (char*) "serial")) {
		file = fopen(fileName, "a");
	}else {
		file = fopen(fileName, "a");
	}
        long double timeTaken = (long double)(end - start)/CLOCKS_PER_SEC;
        fprintf(file, "%s, %d, %d, %d, %Lf\n", whichProgramToRun, users, attributes, k, timeTaken);
        fclose(file);
	printf("%s, %d, %d, %d, %Lf\n", whichProgramToRun, users, attributes, k, timeTaken);
}

void calculateKNearestSerial(int * scores, int * kNearest, int users, int K) {
        int minValue, minIndex, value, user, k, index;
        for(user = 0; user < users; user++) {
                for(k = 0; k < K; k++) {
                        minValue = MAX_VALUE;
                        minIndex = -1;
                        for(index = 0; index < users; index++) {
                                value = scores[user*users + index];
                                if(value < minValue && index != user) {
                                        minValue = value;
                                        minIndex = index;
                                }
                        }
                        if(minIndex != -1) {
                                // scores[user*users + minIndex] = MAX_VALUE;
                        }
                        // kNearest[user*users + k] = minIndex;
                }
        }
}

int main(int argc, char **argv) {
	int * dataSet; int * scores; int users; int attributes; int k;
	preliminarySteps(argc, argv, &dataSet, &scores, &users, &attributes, &k);
	
	char* whichProgramToRun = argv[4];
	
	clock_t start = clock();

	printf("k = %d\n", k);
	// printMatrix(dataSet, users, attributes);

	if(checker(whichProgramToRun, (char*) "serial")) {
		// serial
		calculateScores(dataSet, scores, users, attributes);
		free(dataSet);
		int * kNearest = (int*) malloc(sizeof(int) * users * k);
		calculateKNearestSerial(scores, kNearest, users, k);
		free(scores);
		free(kNearest);
	}else if(checker(whichProgramToRun, (char*) "parallel")) {
		// cuda parallel
		launchCalculateScoreKernel(dataSet, scores, users, attributes);
		free(dataSet);
		int * kNearest = (int*) malloc(sizeof(int) * users * k);
                launchCalculateKNearestKernel(scores, kNearest, users, k);
                free(scores);
                free(kNearest);
	}else {
		printf("Enter correct program to run: serial or parallel.\n");
		free(dataSet); free(scores);
		exit(0);
	}	
	
	// printf("Scores:-\n");
	// printMatrix(scores, users, users);

	clock_t end = clock();

	char * fileName = argv[5];
	writeToFile(start, end, whichProgramToRun, users, attributes, k, fileName);

	return 0;
}
