#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define MAX_VALUE 2147483647
#define numThreads 32

void printMatrix(int *matrix, int users, int attributes) {
	// printf("Matrix:-\n");
	for(int i = 0; i < (users * attributes); i++) {
		if(i % attributes == 0 && i != 0) {
			printf("\n%d ", matrix[i]);
		} else {
			printf("%d ", matrix[i]);
		}
	}
	printf("\n");
}

void preliminarySteps(int argc, char** argv, int** dataSetPtr, int** scoresPtr, int* usersPtr, int* attributesPtr) {
    // Check input
    if(argc < 4) {
        printf("Usage: %s <k> <users> <attributes>\n", argv[0]);
        exit(0);
    }
	int * dataSet;
	int k = atoi(argv[1]);
	int users = atoi(argv[2]);
	int attributes = atoi(argv[3]);
	*usersPtr = users;
	*attributesPtr = attributes;

	dataSet = (int *)malloc(sizeof(int) * users * attributes);
    if(dataSet != NULL) {
        printf("Allocated an array for %d users and %d attributes\n", users, attributes);
    } else {
        printf("Couldn't allocate dataSet array, quitting!\n");
        exit(0);
    }

    // Seed the RNG
    srand(time(NULL));
    // Now fill dataSet with some values
    for(int i=0; i < (users * attributes); i++)
        //dataSet[i] = rand() % 1000; // Random integers between 0 and 1,000
        dataSet[i] = rand() % 15; // Random integers between 0 and 1,000
    *dataSetPtr = dataSet;
    int *scores;
    scores = (int *)malloc(sizeof(int) * users * users);
    if(scores != NULL) {
        printf("Allocated a square scores array for %d users\n", users);
    } else {
        printf("Couldn't allocate scores array, quitting!\n");
        exit(0);
    }
    *scoresPtr = scores;
}

void calculateScore(int* matrix, int* scores, int users, int attributes, int user1, int user2) {
	int answer = 0;
	int user1Start = attributes*user1;
	int user1End = user1Start + attributes - 1;
	int user2Start = attributes*user2;
	int user2End = user2Start + attributes - 1;
	
	int i; int j; int difference;
	for(i = user1Start, j = user2Start; i <= user1End && j <= user2End ; i++, j++) {
		difference = matrix[i] - matrix[j];
		answer += difference*difference;
	}
	
	scores[user1*users + user2] = answer;
}

void calculateScores(int *matrix, int *scores, int users, int attributes) {
	int user1; int user2;
	for(user1 = 0; user1 < users; user1++) {
		for(user2 = 0; user2 < users; user2++) {
			calculateScore(matrix, scores, users, attributes, user1, user2);
		}
	}
}

__global__ void calculateScoreKernel(int *matrix, int *scores, int users, int attributes) {
        int user1 = numThreads*blockIdx.x + threadIdx.x;
	int user2 = numThreads*blockIdx.y + threadIdx.y;
	
        int answer = 0;
        int user1Start = attributes*user1;
        int user1End = user1Start + attributes - 1;
        int user2Start = attributes*user2;
        int user2End = user2Start + attributes - 1;

        int i; int j; int difference;
        for(i = user1Start, j = user2Start; i <= user1End && j <= user2End ; i++, j++) {
                difference = matrix[i] - matrix[j];
                answer += difference*difference;
        }

        scores[user1*users + user2] = answer;
}

void launchCalculateScoreKernel(int * dataSet, int * scores, int users, int attributes) {
	int * dev_dataSet;
	int * dev_scores;
	
	hipMalloc((void**) &dev_dataSet, users*attributes*sizeof(int));
	hipMalloc((void**) &dev_scores, users*users*sizeof(int));
	
	hipMemcpy(dev_dataSet, dataSet, users*attributes*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_scores, scores, users*users*sizeof(int), hipMemcpyHostToDevice);

	int numBlocks = (int) ceil(users*1.0/numThreads);
	dim3 grid( numBlocks, numBlocks, 1 );
	dim3 block( numThreads, numThreads, 1 );
	calculateScoreKernel<<< grid, block >>>(dev_dataSet, dev_scores, users, attributes);

	hipMemcpy(scores, dev_scores, users*users*sizeof(int), hipMemcpyDeviceToHost);
}

int main(int argc, char **argv) {
	int * dataSet; int * scores; int users; int attributes;
	preliminarySteps(argc, argv, &dataSet, &scores, &users, &attributes);
	
	printf("Matrix:-\n");
	printMatrix(dataSet, users, attributes);

	// serial
	// calculateScores(dataSet, scores, users, attributes);

	launchCalculateScoreKernel(dataSet, scores, users, attributes);	
	
	printf("Scores:-\n");
	printMatrix(scores, users, users);
	
	// Clean up after ourselves
	free(dataSet); free(scores);
	return 0;
}
