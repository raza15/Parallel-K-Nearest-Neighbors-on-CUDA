
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define MAX_VALUE 2147483647

const int USERS = 3;
const int ATTRIBUTES = 5;
// const int K = 2;

void readDataFromFile(const char * fileName, int matrix[USERS][ATTRIBUTES]) {
	FILE * fp;  
        fp = fopen(fileName, "r+");
        int row; int col;
        for(row = 0; row < USERS; row++) {
                for(col = 0; col < ATTRIBUTES; col++) {
                        fscanf(fp, "%d", &matrix[row][col]);
                }
        }
}

void printMatrix(int matrix[USERS][ATTRIBUTES]) {
	int row; int col;
	printf("Matrix:-\n");
	for(row = 0; row < USERS; row++) {
		for(col = 0; col < ATTRIBUTES; col++) {
			printf("%d, ", matrix[row][col]);
		}
		printf("\n");
	}
}

void printArray(int * array, int size) {
	int i;
	for(i = 0; i < size; i++) {
		printf("%d, ", array[i]);
	}
	printf("\n");
}

int* matrixTo1DArray(int matrix[USERS][ATTRIBUTES]) {
	int* newArray = new int[USERS*ATTRIBUTES];
	int h; int w;
	for(h = 0; h < USERS; h++) {
		for(w = 0; w < ATTRIBUTES; w++) {
			newArray[ATTRIBUTES * h + w] = matrix[h][w];
		}
	}
	return newArray;
}
/*
__global__ void calculateScoresKernel(int * matrixArray, int * scores) {
	int row; int col;
	int row1[ATTRIBUTES];
	int row2[ATTRIBUTES];
	for(row = 0; row < USERS; row++) {
		copyAllAttributes(matrix[row], row1);
		for(col = 0; col < USERS; col++) {
			copyAllAttributes(matrix[col], row2);
			int distance;
			eucladeanDistance(row1, row2, &distance);
			scores[row][col] = distance;
		}
	}
}
*/
int* calculateScores(int * array) {
	int * output = new int[USERS*USERS];
	return output;
}

int main(void) {
	int matrix[USERS][ATTRIBUTES];
	readDataFromFile("testData.txt", matrix);
	printMatrix(matrix);
	int * matrixArray = matrixTo1DArray(matrix);
	// printArray(matrixArray, USERS*ATTRIBUTES);
	int * scores = calculateScores(matrixArray);
}
